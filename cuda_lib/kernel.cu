#include "hip/hip_runtime.h"
__global__
void apply_instruction_set_to_reference_seq_gpu(char* res_array, char* ref_stream, char* alt_Stream, 
    size_t* exc_code, size_t* start_pos, size_t* length, size_t* start_pos_res, unsinged int int num_task)
{
    int thread_index=(blockIdx.x * blockDim.x) + threadIdx.x; // get the thread index
	int step_size = blockDim.x + gridDim.x; // the step size

    for(int work_idx=thread_index; work_idx < num_task; work_idx +=step_size)
    { 
        if(exc_code[work_idx]==0) // A copy based instruction
        {
        // apply the copy based instruction for the whole length of the instruction
            for(int ins_count=0; ins_count!=length[work_idx];ins_count++)
            {
                res_array[start_pos_res[work_idx]+ins_count]=ref_stream[start_pos[work_idx]+ins_count]; 
            }
        }
        else // an insertion based instruction
        {
            for(int ins_count=0; ins_count!=instruction_length[work_idx];ins_count++)
            {
                res_array[start_pos_res[work_idx]+ins_count]=alt_Stream[start_pos[work_idx]+ins_count];  
            }
        }
    }

/** C wrapper */
extern "C"
{  
    /*@brief: A C wrapper for the CUDA kernel that perform Task execution 
    * @param ref_seq: A pointer to a char array the will hold the results 
    * @param ref_seq: A pointer to a char array holding the reference stream 
    * @param alt_stream: A pointer to a char array holding the alteration stream 
    * @param exc_code: A pointer to a size_t array holding the execution code results 
    * @param start_pos: A pointer to a size_t array holding the start position in the input stream, whether it is the ref or alternative
    * @param length: A pointer to a size_t array holding the length of the instruction 
    * @param start_pos_ref: A pointer to a size_t array holding the position in the resulting array  
    * @param num_task: the number of tasks which equal the length of the following arrays: ref_seq, ref_stream, alt_stream and exc_code
    * @param len_res_array: the length of the results array
    * @param len_ref_stream: the length of the refernce stream 
    * @param len_alt_stream: the length of the alt stream
    * @Notes: Error code meaning: 
    *       1. 0 => Sucess 
    *       2. 1 => GPU allocation failure 
    *       3. 2 => Failure with copying the data to the GPU 
    *       4. 3 => Launching the kernel failed 
    *       5. 4 => Kernel execution failed
    *       6. 5 => Copying the results array to the host failed 
    */
    int kernel_wrapper(char* res_array, char* ref_stream, char* alt_stream, size_t* exc_code,
        size_t* start_pos, size_t* length, size_t* start_pos_res, size_t num_task, 
        size_t len_res_array, size_t len_ref_stream,
        size_t len_alt_stream) 
    {
        // allocate arrays on the GPU
        //---------------------------
        // 1. creating pointers:
        //----------------------
        char* res_array_ptr; 
        char* ref_stream_ptr; 
        char* alt_stream_ptr; 
        size_t* exc_code_ptr; 
        size_t* start_pos_ptr; 
        size_t* length_ptr; 
        size_t* start_pos_res_ptr; 
        // 2. Perform allocations 
        //-----------------------
        if(hipMalloc(&res_array_ptr,len_res_array*sizeof(char))!=hipSuccess)return 1;
        if(hipMalloc(&ref_stream_ptr,len_ref_stream*sizeof(char))!=hipSuccess)return 1; 
        if(hipMalloc(&alt_stream_ptr,len_alt_stream*sizeof(char))!=hipSuccess)return 1; 
        if(hipMalloc(&exc_code_ptr,num_task*sizeof(size_t))!=hipSuccess)return 1; 
        if(hipMalloc(&start_pos_ptr,num_task*sizeof(size_t))!=hipSuccess)return 1; 
        if(hipMalloc(&length_ptr,num_task*sizeof(size_t))!=hipSuccess)return 1; 
        if(hipMalloc(&start_pos_res_ptr,num_task*sizeof(size_t))!=hipSuccess)return 1; 
        // 3. Copy the data to the GPU
        //---------------------------- 
        if(hipMemcpy(ref_stream_ptr, ref_stream, len_ref_stream*sizeof(char),hipMemcpyHostToDevice)!=hipSuccess)return 2;
        if(hipMemcpy(alt_stream_ptr, alt_stream, len_alt_stream*sizeof(char),hipMemcpyHostToDevice)!=hipSuccess)return 2;
        if(hipMemcpy(exc_code_ptr, exc_code, num_task*sizeof(size_t),hipMemcpyHostToDevice)!=hipSuccess)return 2;
        if(hipMemcpy(start_pos_ptr, start_pos, num_task*sizeof(size_t),hipMemcpyHostToDevice)!=hipSuccess)return 2;
        if(hipMemcpy(length_ptr, length, num_task*sizeof(size_t),hipMemcpyHostToDevice)!=hipSuccess)return 2;
        if(hipMemcpy(start_pos_res_ptr, start_pos_r, num_task*sizeof(size_t),hipMemcpyHostToDevice)!=hipSuccess)return 2;
        // Launching the kernel 
        //----------------------
        unsigned int num_threads_per_block=1024; 
        unsigned int number_blocks=(num_task/num_threads_per_block) +1 ; 
        apply_instruction_set_to_reference_seq_gpu<<<number_blocks,num_threads_per_block>>>
        (
            res_array_ptr, ref_stream_ptr, alt_stream_ptr, exc_code_ptr, 
            start_pos_ptr, length_ptr, start_pos_res_ptr, num_task 
        );         
        if(hipGetLastError()!=hipSuccess)return 3;
        // Synchronize the calling code: 
        //------------------------------
        cudaErr_t err=hipDeviceSynchronize(); 
        if (err!=hipSuccess) return 4; 
        // Copying the from the GPU 
        //-------------------------
        if(hipMemcpy(res_array, res_array_ptr, len_res_array*sizeof(char), hipMemcpyDeviceToHost)!=hipSuccess) return 5;
        // Relase allocated on the GPU 
        //----------------------------
        hipFree(res_array_ptr); 
        hipFree(ref_stream_ptr); 
        hipFree(alt_stream_ptr); 
        hipFree(exc_code_ptr); 
        hipFree(start_pos_ptr); 
        hipFree(length_ptr); 
        hipFree(start_pos_res_ptr);
        // return a success state
        //------------------------
        return 0;         
    }
} 